#include "hip/hip_runtime.h"
#include "stencil.h"

#define NUM_BLOCKS 512
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

using namespace std;

__global__
void stencilKernel (float *a, float *c) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int lid = threadIdx.x; // local thread id within a block
	
	__shared__ float temp[NUM_THREADS_PER_BLOCK + 4];
	temp[lid+2] = a[id];
	if (lid == 0 && id != 0) {
		printf("id = %d\n", id);
		temp[lid] = a[id-2];
		temp[lid+1] = a[id-1];
	}
	if (lid == NUM_THREADS_PER_BLOCK - 1 && id != SIZE - 1) {
		printf("id = %d\n", id);
		temp[lid+3] = a[id+1];
		temp[lid+4] = a[id+2];
	}

	__syncthreads(); // wait for all threads within a block
	c[id] = 0;
	for (int n = -2; n <= 2; n++) {
		if ((id + n >= 0) && (id + n < SIZE)) {
			// c[id] += a[id + n];
			c[id] += temp[lid + 2 + n];
		}
	}
}

void stencil (float *a, float *c) {
	chrono::steady_clock::time_point begin = chrono::steady_clock::now();

	for (int i = 0; i < SIZE; i++) {
		// considers 4 neighbours
		for (int n = -2; n <= 2; n++) {
			if ((i + n >= 0) && (i + n < SIZE))
				c[i] += a[i + n];
		}
	}

	chrono::steady_clock::time_point end = chrono::steady_clock::now();
	cout << endl << "Sequential CPU execution: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl << endl;
}

void launchStencilKernel (float *a, float *c) {
	// pointers to the device memory
	float *da, *dc;
	// declare variable with size of the array in bytes
	int bytes = SIZE * sizeof(float);

	// allocate the memory on the device
	hipMalloc ((void**) &da, bytes);
	hipMalloc ((void**) &dc, bytes);
	checkCUDAError("mem allocation");

	// copy inputs to the device
	hipMemcpy (da, a, bytes, hipMemcpyHostToDevice);
	checkCUDAError("memcpy h->d");

	// launch the kernel
	startKernelTime ();
	stencilKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (da, dc);
	stopKernelTime ();
	checkCUDAError("kernel invocation");

	// copy the output to the host
	hipMemcpy (c, dc, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");

	// free the device memory
	hipFree(da);
	hipFree(dc);
	checkCUDAError("mem free");
}

int main( int argc, char** argv) {
	// arrays on the host
	float a[SIZE], b[SIZE], c[SIZE];

	// initialises the array
	for (unsigned i = 0; i < SIZE; ++i)
		a[i] = (float) rand() / RAND_MAX;

	stencil (a, b);
	
	launchStencilKernel (a, c);

	return 0;
}
